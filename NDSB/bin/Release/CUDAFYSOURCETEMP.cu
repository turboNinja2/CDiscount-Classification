
#include <hip/hip_runtime.h>

// NDSB.Stats
extern "C" __global__  void GetDistanceToClasses(int n, int p,  float* trainLines, int trainLinesLen0, int trainLinesLen1,  float* currentLine, int currentLineLen0,  float* result, int resultLen0);

// NDSB.Stats
extern "C" __global__  void GetDistanceToClasses(int n, int p,  float* trainLines, int trainLinesLen0, int trainLinesLen1,  float* currentLine, int currentLineLen0,  float* result, int resultLen0)
{
	int x = threadIdx.x;
	int num = blockIdx.x * blockDim.x + x;
	if (num > n)
	{
		return;
	}
	float num2 = 0.0f;
	for (int i = 0; i < p; i++)
	{
		float num3 = trainLines[(num) * trainLinesLen1 + ( i)];
		float num4 = currentLine[(i)];
		num2 += (num3 - num4) * (num3 - num4);
	}
	result[(num)] = num2;
}
